#include "hip/hip_runtime.h"
#include "amr.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#define max(x, y) ((x) > (y) ? (x) : (y))
#define min(x, y) ((x) < (y) ? (x) : (y))
#define abs(x) (x < 0 ? -(x) : x)

double AFFECT_RATE;
double EPSILON;
int boxCount;
Box *boxes;
double maxTemp, minTemp;
double timeDevice;

int hasConverged() 
{
    maxTemp = minTemp = boxes[0].temp;

    for (int i = 1; i < boxCount; i++) 
    {
        if (boxes[i].temp > maxTemp) 
        {
            maxTemp = boxes[i].temp;
        }

        if (boxes[i].temp < minTemp) 
        {
            minTemp = boxes[i].temp;
        }
    }

    return maxTemp - minTemp > maxTemp * EPSILON ? 0 : 1;
}

void freeMemory() 
{
    int i, j;

    for (i = 0; i < boxCount; i++) 
    {
        for (j = 0; j < 4; j++) 
        {
            free(boxes[i].neighborsInDir[j]);
        }

        free(boxes[i].allNeighbors);
    }

    free(boxes);
}

void getNeighborConfig(Box *box) 
{
    box->totalNeighborsCount = 0;
    for (int i = 0; i < 4; i++)
    {
        box->totalNeighborsCount += box->neighborCountInDir[i];
    }
    box->allNeighbors = (Neighbor *) malloc(box->totalNeighborsCount * sizeof(Neighbor));
    int k = 0;
    box->uncommonEdgeLength = box->perimeter;
    for (int i = 0; i < 4; i++)
    {
        for (int j = 0; j < box->neighborCountInDir[i]; j++)
        {
            Box neighbor = boxes[box->neighborsInDir[i][j]];
            int commonEdge = 0;
            if (i < 2) {
                commonEdge = abs(max(box->upperLeftX, neighbor.upperLeftX) - min(box->upperLeftX + box->width, neighbor.upperLeftX + neighbor.width));
            } else {
                commonEdge = abs(max(box->upperLeftY, neighbor.upperLeftY) - min(box->upperLeftY + box->height, neighbor.upperLeftY + neighbor.height));
            }
            box->allNeighbors[k].id = neighbor.id;
            box->allNeighbors[k].commonEdgeLength = commonEdge;
            box->uncommonEdgeLength -= commonEdge;
            k++;
        }
        
    }
    
}

int getBoxConfig(Box *box) 
{
    if(!scanf("%d", &box->id)) 
    {
        return 0;
    }
    if(!scanf("%d %d %d %d", &box->upperLeftY, &box->upperLeftX, &box->height, &box->width)) 
    {
        return 0;
    }
    box->perimeter = 2*(box->height + box->width);

    for (int i = 0; i < 4; i++)
    {
        if(!scanf("%d", &box->neighborCountInDir[i])) 
        {
            return 0;
        }
        box->neighborsInDir[i] = (int *) malloc(box->neighborCountInDir[i] * sizeof(int));
        for (int j = 0; j < box->neighborCountInDir[i]; j++)
        {
            if(!scanf("%d", &box->neighborsInDir[i][j])) 
            {
                return 0;
            }
        }
    }
    if(!scanf("%lf", &box->temp)) 
    {
        return 0;
    }
    return 1;
}

int getInput() 
{
    if(!scanf("%d %*d %*d", &boxCount)) 
    {
        return 0;
    }

    boxes = (Box *) malloc(boxCount * sizeof(Box));
    for (int i = 0; i < boxCount; i++)
    {
        getBoxConfig(&boxes[i]);
    }

    int garbage = scanf("%*d");
    return 1;
}


__global__ void calcNewTemp(Box *dboxes, double *dtemp, double affectRate, int boxCnt) 
{   
    int boxId = blockIdx.x*blockDim.x + threadIdx.x;
    if (boxId < boxCnt) 
    {
        double waat = dboxes[boxId].temp * dboxes[boxId].uncommonEdgeLength;
        for (int i = 0; i < dboxes[boxId].totalNeighborsCount; i++) 
        {
            waat += dboxes[boxId].allNeighbors[i].commonEdgeLength * dboxes[dboxes[boxId].allNeighbors[i].id].temp;
        }
        waat /= dboxes[boxId].perimeter;
        dtemp[boxId] = dboxes[boxId].temp + (waat - dboxes[boxId].temp) * affectRate;
    }
}


__global__ void updateTemp(Box *dboxes, double *dtemp, int boxCnt) 
{   
    int boxId = blockIdx.x*blockDim.x + threadIdx.x;
    if(boxId < boxCnt) 
    {
        dboxes[boxId].temp = dtemp[boxId];
    }
}


int runConvergenceLoopDevice() 
{
    int iteration,j;
    
    //Memory allocation
    Box *deviceBoxes;
    Neighbor *dboxNeighbors[boxCount];
    double *newTemp, *newDeviceTemp;
    int temp_memsize = boxCount * sizeof(double);
    int boxes_memsize = boxCount * sizeof(Box);
    newTemp = (double *) malloc(temp_memsize);
    hipMalloc((void **) &newDeviceTemp, temp_memsize);
    hipMalloc((void **) &deviceBoxes, boxes_memsize);
    for (int i=0; i<boxCount; i++)
    {
        hipMalloc((void **) &dboxNeighbors[i], boxes[i].totalNeighborsCount * sizeof(Neighbor));
    }
    int thread_per_block = 128;
    int nBlocks = (boxCount + thread_per_block -1)/thread_per_block;
    dim3 dimGrid(nBlocks);
    dim3 dimBlock(thread_per_block);
    hipMemcpy(deviceBoxes, boxes, boxes_memsize, hipMemcpyHostToDevice);
    for (int i=0; i<boxCount; i++)
    {
        hipMemcpy(dboxNeighbors[i], boxes[i].allNeighbors, boxes[i].totalNeighborsCount * sizeof(Neighbor), hipMemcpyHostToDevice);
        hipMemcpy(&(deviceBoxes[i].allNeighbors), &dboxNeighbors[i], sizeof(Neighbor*), hipMemcpyHostToDevice);
    }

    for (iteration = 0; !hasConverged(); iteration++) 
    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        calcNewTemp<<<dimGrid, dimBlock>>>(deviceBoxes, newDeviceTemp, AFFECT_RATE, boxCount);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float diff = 0;
        hipEventElapsedTime(&diff, start, stop);
        timeDevice += diff/1000;

        updateTemp<<<dimGrid, dimBlock>>>(deviceBoxes, newDeviceTemp, boxCount);
        hipMemcpy(newTemp, newDeviceTemp, temp_memsize, hipMemcpyDeviceToHost);
        for (j = 0; j < boxCount; j++) 
        {
             boxes[j].temp = newTemp[j];
        }
    }

    free(newTemp);
    hipFree(newDeviceTemp);
    hipFree(deviceBoxes);
    hipFree(dboxNeighbors[boxCount]);
    return iteration;
}


int main(int argc, char **argv) 
{
    sscanf(argv[1], "%lf", &AFFECT_RATE);
    sscanf(argv[2], "%lf", &EPSILON);
    if(!getInput()) 
    {
        printf("Error in input!");
    }

    for (int i = 0; i < boxCount; i++)
    {
        getNeighborConfig(&boxes[i]);
    }

    int flopsPerIteration = 0;
    for(int i=0; i<boxCount; i++)
    {
        flopsPerIteration += boxes[i].totalNeighborsCount*2+5;
    }

    int iterationsDevice = runConvergenceLoopDevice();
    double nGigaFlop = iterationsDevice * (flopsPerIteration / 1000000000.0);
    printf("Number of flop per iteration: %d\n", flopsPerIteration);
    printf("Total number of Giga flop in Device: %lf\n", nGigaFlop);

    printf("\n**************************************************************************\n");
    printf("dissipation converged in %d iterations,\n", iterationsDevice);
    printf("    with max DSV = %.7lf and min DSV = %.7lf\n", maxTemp, minTemp);
    printf("    affect rate  = %lf; epsilon = %lf\n\n", AFFECT_RATE, EPSILON);
    double GigaFlopsPerSec = nGigaFlop / timeDevice;
    printf("Time taken on device (ms) = %lf\n", timeDevice * 1000);
    printf("Giga FLOPS per sec on device = %lf\n", GigaFlopsPerSec);
    printf("\n**************************************************************************\n");

    freeMemory();
    hipDeviceReset();
    return 0;
}

